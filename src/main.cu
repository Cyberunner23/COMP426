#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <thread>
#include <vector>

#include "BallUtils.hpp"
#include "GLUtils.hpp"

//*********************************************************
// Constants
//*********************************************************
const unsigned int FrameRate = 30;
const float FrameTime = 1.0f / FrameRate;


static void error_callback(int error, const char* description)
{
    std::cout << "[ERROR][GLFW]: " << description << std::endl;
}

double do_frame_rate_limiting(double& lastFrameStartTime)
{
    double currentTime = glfwGetTime();

    double deltaTime = currentTime - lastFrameStartTime;
    if (deltaTime < FrameTime)
    {
        std::this_thread::sleep_for(std::chrono::milliseconds((unsigned int)((FrameTime - deltaTime) * 1000)));
    }

    currentTime = glfwGetTime();
    double deltaT = currentTime - lastFrameStartTime;

    lastFrameStartTime = glfwGetTime();

    return deltaT;
}

void display_background()
{
    for (int x = 0; x < 4; ++x)
    {
        for (int y = 0; y < 4; ++y)
        {
            float posX = ((float)(2 * x) + 1) * (float)WinSize / 8;
            float posY = ((float)(2 * y) + 1) * (float)WinSize / 8;
            DrawGradientSquare(glm::vec2{posX, posY});
        }
    }
}

void display_circles(std::vector<BallState> balls)
{
    for (BallState ball : balls)
    {
        DrawCircle(ball.Position, ball.Color, ball.Radius);
    }
}

void handle_wall_collision(BallState& ball)
{
    if (collides_with_edge_x(ball))
    {
        float vX = ball.Velocity.x;
        ball.Velocity.x = -vX;
    }

    if (collides_with_edge_y(ball))
    {
        float vY = ball.Velocity.y;
        ball.Velocity.y = -vY;
    }
}

void handle_ball_ball_collision(BallState& b1, BallState& b2)
{

    glm::vec2 delta = b1.Position - b2.Position;
    float r = (float)b1.Radius + (float)b2.Radius;
    float dist2 = glm::dot(delta, delta);

    if (dist2 >= r * r)
    {
        return;
    }

    float d = glm::length(delta);

    glm::vec2 mtd;
    if (d != 0.0f)
    {
        mtd = delta * ((((float)b1.Radius + (float)b2.Radius) - d)/d);
    }
    else
    {
        d = (float)b1.Radius +  (float)b2.Radius - 1.0f;
        delta = glm::vec2 {b1.Radius + b2.Radius, 0.0f};
        mtd = delta * ((((float)b1.Radius + (float)b2.Radius) - d)/d);
    }

    float im1 = 1 / b1.Mass; // inverse mass quantities
    float im2 = 1 / b2.Mass;

    b1.Position = b1.Position + (mtd * (im1 / (im1 + im2)));
    b2.Position = b2.Position - (mtd * (im1 / (im1 + im2)));

    glm::vec2 v = (b1.Velocity - b2.Velocity);
    float vn = glm::dot(v, glm::normalize(mtd));

    if (vn > 0.0f) return;

    float i = (-(1.0f + 0.85f) * vn) / (im1 + im2);
    glm::vec2 impulse = mtd * i * 0.001f;

    b1.Velocity = b1.Velocity + (impulse * im1);
    b2.Velocity = b2.Velocity - (impulse * im2);
}

void handle_collisions(std::vector<BallState>& balls)
{
    for (int i = 0; i < balls.size(); ++i)
    {
        handle_wall_collision(balls[i]);

        for(int j = i + 1; j < balls.size(); j++)
        {
            handle_ball_ball_collision(balls[i], balls[j]);
        }
    }
}

__global__ void update_ball_position(BallState* balls, double deltaT)
{
    auto id = threadIdx.x;
    update_ball(balls[id], deltaT);
}

int main(int argc, char **argv)
{
    GLFWwindow* window;

    glfwSetErrorCallback(error_callback);
    if (!glfwInit())
    {
        std::cout << "[ERROR][GLFW]: Failed to init GLFW" << std::endl;
        return -1;
    }

    window = glfwCreateWindow(WinSize, WinSize, "COMP 426 A1", nullptr, nullptr);
    if (!window)
    {
        glfwTerminate();
        std::cout << "[ERROR][GLFW]: Failed to create GLFW window" << std::endl;
        return -1;
    }

    glfwMakeContextCurrent(window);

    // Host
    std::vector<BallState> balls = initialize_balls(argc, argv);
    auto count = balls.size();
    auto memorySize = count * sizeof(BallState);
    // Device
    std::cout << memorySize << std::endl;
    BallState* deviceBalls;
    hipMalloc(&deviceBalls, memorySize);

    double lastFrameStartTime = glfwGetTime();
    while(!glfwWindowShouldClose(window))
    {
        double deltaT = do_frame_rate_limiting(lastFrameStartTime);

        glClear(GL_COLOR_BUFFER_BIT);
        glEnable(GL_BLEND);
        glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

        // CopyHost to GPU
        hipMemcpy(deviceBalls, balls.data(), memorySize, hipMemcpyHostToDevice);

        int blockSize = count;
        int gridSize = 1;
        update_ball_position<<<gridSize, blockSize>>>(deviceBalls, deltaT);
        hipMemcpy(balls.data(), deviceBalls, memorySize, hipMemcpyDeviceToHost);

        handle_collisions(balls);

        display_background();
        display_circles(balls);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    hipFree(deviceBalls);

    return 0;
}